#include "hip/hip_runtime.h"
/*
 * -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * Acknowledgements: This example is based on cvAdvDiff_bnd 
 *                   example by Scott D. Cohen, Alan C. 
 *                   Hindmarsh and Radu Serban @ LLNL
 * -----------------------------------------------------------------
 * Example problem:
 *
 * The following is a simple example problem with a banded Jacobian,
 * with the program for its solution by CVODE.
 * The problem is the semi-discrete form of the advection-diffusion
 * equation in 2-D:
 *   du/dt = d^2 u / dx^2 + .5 du/dx + d^2 u / dy^2
 * on the rectangle 0 <= x <= 2, 0 <= y <= 1, and the time
 * interval 0 <= t <= 1. Homogeneous Dirichlet boundary conditions
 * are posed, and the initial condition is
 *   u(x,y,t=0) = x(2-x)y(1-y)exp(5xy).
 * The PDE is discretized on a uniform MX+2 by MY+2 grid with
 * central differencing, and with boundary values eliminated,
 * leaving an ODE system of size NEQ = MX*MY.
 * This program solves the problem with the BDF method, Newton
 * iteration with the CVBAND band linear solver, and a user-supplied
 * Jacobian routine.
 * It uses scalar relative and absolute tolerances.
 * Output is printed at t = .1, .2, ..., 1.
 * Run statistics (optional outputs) are printed at the end.
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <cvode/cvode.h>
#include <sunlinsol/sunlinsol_spgmr.h> /* access to SPGMR SUNLinearSolver        */
#include <cvode/cvode_spils.h>         /* access to CVSpils interface            */
#include <nvector/nvector_raja.h>
#include <sundials/sundials_types.h>
#include <sundials/sundials_math.h>

#include <RAJA/RAJA.hpp>


/* Real Constants */

#define ATOL  RCONST(1.0e-5) /* scalar absolute tolerance */
#define T0    RCONST(0.0)    /* initial time              */
#define T1    RCONST(0.1)    /* first output time         */
#define DTOUT RCONST(0.1)    /* output time increment     */
#define NOUT  10             /* number of output times    */

#define ZERO RCONST(0.0)
#define HALF RCONST(0.5)
#define ONE  RCONST(1.0)
#define TWO  RCONST(2.0)
#define FIVE RCONST(5.0)


/* Type : _UserData (contains model and discretization parameters) */
struct _UserData {
  sunindextype MX, MY, NEQ;
  realtype dx, dy, XMAX, YMAX;
  realtype hdcoef, hacoef, vdcoef;
};

typedef _UserData *UserData;

/* Problem setup and initialization functions */
static UserData SetUserData(int argc, char** argv);
static void SetIC(N_Vector u, UserData data);

/* Functions Called by the Solver */
static int f(realtype t, N_Vector u, N_Vector udot, void *user_data);
static int jtv(N_Vector v, N_Vector Jv, realtype t,
               N_Vector u, N_Vector fu,
               void *user_data, N_Vector tmp);

/* Private Helper Functions */
static void PrintHeader(realtype reltol, realtype abstol, realtype umax, UserData data);
static void PrintOutput(realtype t, realtype umax, long int nst);
static void PrintFinalStats(void *cvode_mem);

/* Private function to check function return values */
static int check_flag(void *flagvalue, const char *funcname, int opt);


/*
 *-------------------------------
 * Main Program
 *-------------------------------
 */

int main(int argc, char** argv)
{
  realtype reltol, abstol, t, tout, umax;
  N_Vector u;
  UserData data;
  SUNLinearSolver LS;
  void *cvode_mem;
  int iout, flag;
  long int nst;

  u = NULL;
  data = NULL;
  LS = NULL;
  cvode_mem = NULL;

  /* Set model parameters */
  data = SetUserData(argc, argv);
  if(check_flag((void *)data, "malloc", 2)) return(1);

  /* Set the tolerances */
  reltol = ZERO;
  abstol = ATOL;

  /* Create a RAJA vector with initial values */
  u = N_VNew_Raja(data->NEQ);  /* Allocate u vector */
  if(check_flag((void*)u, "N_VNew_Raja", 0)) return(1);

  SetIC(u, data);  /* Initialize u vector */

  /* Call CVodeCreate to create the solver memory and specify the 
   * Backward Differentiation Formula and the use of a Newton iteration */
  cvode_mem = CVodeCreate(CV_BDF, CV_NEWTON);
  if(check_flag((void *)cvode_mem, "CVodeCreate", 0)) return(1);

  /* Call CVodeInit to initialize the integrator memory and specify the
   * user's right hand side function in u'=f(t,u), the initial time T0, and
   * the initial dependent variable vector u. */
  flag = CVodeInit(cvode_mem, f, T0, u);
  if(check_flag(&flag, "CVodeInit", 1)) return(1);

  /* Call CVodeSStolerances to specify the scalar relative tolerance
   * and scalar absolute tolerance */
  flag = CVodeSStolerances(cvode_mem, reltol, abstol);
  if (check_flag(&flag, "CVodeSStolerances", 1)) return(1);

  /* Set the pointer to user-defined data */
  flag = CVodeSetUserData(cvode_mem, data);
  if(check_flag(&flag, "CVodeSetUserData", 1)) return(1);

  /* Create SPGMR solver structure without preconditioning
   * and the maximum Krylov dimension maxl */
  LS = SUNSPGMR(u, PREC_NONE, 0);
  if(check_flag(&flag, "SUNSPGMR", 1)) return(1);

  /* Set CVSpils linear solver to LS */
  flag = CVSpilsSetLinearSolver(cvode_mem, LS);
  if(check_flag(&flag, "CVSpilsSetLinearSolver", 1)) return(1);

  /* Set the Jacobian-times-vector function */
  flag = CVSpilsSetJacTimes(cvode_mem, NULL, jtv);
  if(check_flag(&flag, "CVSpilsSetJacTimesVecFn", 1)) return(1);

  /* In loop over output points: call CVode, print results, test for errors */

  umax = N_VMaxNorm(u);
  PrintHeader(reltol, abstol, umax, data);
  for(iout=1, tout=T1; iout <= NOUT; iout++, tout += DTOUT) {
    flag = CVode(cvode_mem, tout, u, &t, CV_NORMAL);
    if(check_flag(&flag, "CVode", 1)) break;
    umax = N_VMaxNorm(u);
    flag = CVodeGetNumSteps(cvode_mem, &nst);
    check_flag(&flag, "CVodeGetNumSteps", 1);
    PrintOutput(t, umax, nst);
  }

  PrintFinalStats(cvode_mem);  /* Print some final statistics   */

  N_VDestroy(u);          /* Free the u vector */
  CVodeFree(&cvode_mem);  /* Free the integrator memory */
  free(data);             /* Free the user data */

  return(0);
}

/*
 *-------------------------------------------
 * Problem setup and initialization functions
 *-------------------------------------------
 */

/* Set model and discretization parameters */

UserData SetUserData(int argc, char *argv[])
{
  const sunindextype MX = 10;
  const sunindextype MY = 5;
  const realtype XMAX = RCONST(2.0);    /* domain boundaries         */
  const realtype YMAX = RCONST(1.0);

  /* Allocate user data structure */
  UserData ud = (UserData) malloc(sizeof *ud);
  if(check_flag((void*) ud, "AllocUserData", 2)) return(NULL);

  ud->MX  = MX;
  ud->MY  = MY;
  ud->NEQ = MX*MY;
  ud->XMAX = XMAX;
  ud->YMAX = YMAX;
  ud->dx = XMAX/(MX+1);  /* Set grid coefficients in data */
  ud->dy = YMAX/(MY+1);
  ud->hdcoef = ONE/(ud->dx*ud->dx);
  ud->hacoef = HALF/(TWO*ud->dx);
  ud->vdcoef = ONE/(ud->dy*ud->dy);

  return ud;
}


/* Set initial conditions in u vector */

static void SetIC(N_Vector u, UserData data)
{
  /* Extract needed constants from data */

  const realtype dx = data->dx;
  const realtype dy = data->dy;
  const realtype xmax = data->XMAX;
  const realtype ymax = data->YMAX;
  const sunindextype MY = data->MY;
  const sunindextype NEQ = data->NEQ;

  /* Extract pointer to solution vector data on the host */
  realtype *udata = N_VGetHostArrayPointer_Raja(u);

  sunindextype i, j, tid;
  realtype x, y;


  /* Load initial profile into u vector */

  for (tid=0; tid < NEQ; tid++) {
    i = tid / MY;
    j = tid % MY;

    x = (i+1)*dx;
    y = (j+1)*dy;

    udata[tid] = x*(xmax - x)*y*(ymax - y)*SUNRexp(FIVE*x*y);
  }
  N_VCopyToDevice_Raja(u);
}


/*
 *-------------------------------
 * Functions called by the solver
 *-------------------------------
 */

/* f routine. Compute f(t,u). */

static int f(realtype t, N_Vector u, N_Vector udot, void *user_data)
{
  UserData data = (UserData) user_data;

  /* Extract needed constants from data */
  const sunindextype MX  = data->MX;
  const sunindextype MY  = data->MY;
  const sunindextype NEQ = data->NEQ;
  const realtype hordc   = data->hdcoef;
  const realtype horac   = data->hacoef;
  const realtype verdc   = data->vdcoef;

  /* Extract pointers to vector data */
  const realtype *udata = N_VGetDeviceArrayPointer_Raja(u);
  realtype *dudata      = N_VGetDeviceArrayPointer_Raja(udot);

  const sunindextype zero = 0;

  RAJA::forall<RAJA::cuda_exec<256> >(zero, NEQ, [=] __device__(sunindextype index) {
    sunindextype i = index/MY;
    sunindextype j = index%MY;

    realtype uab = udata[index];

    realtype udn = (j == 0)    ? ZERO : udata[index - 1];
    realtype uup = (j == MY-1) ? ZERO : udata[index + 1];
    realtype ult = (i == 0)    ? ZERO : udata[index - MY];
    realtype urt = (i == MX-1) ? ZERO : udata[index + MY];

    /* Set diffusion and advection terms and load into udot */

    realtype hdiff = hordc*(ult -TWO*uab + urt);
    realtype hadv  = horac*(urt - ult);
    realtype vdiff = verdc*(udn -TWO*uab + uup);

    dudata[index] = hdiff + hadv + vdiff;
  });

  return(0);
}


/* Jacobian-times-vector routine. */

static int jtv(N_Vector v, N_Vector Jv, realtype t,
               N_Vector u, N_Vector fu,
               void *user_data, N_Vector tmp)
{
  UserData data = (UserData) user_data;

  /* Extract needed constants from data */
  const sunindextype MX  = data->MX;
  const sunindextype MY  = data->MY;
  const sunindextype NEQ = data->NEQ;
  const realtype hordc   = data->hdcoef;
  const realtype horac   = data->hacoef;
  const realtype verdc   = data->vdcoef;

  /* Extract pointers to vector data */
  const realtype *vdata = N_VGetDeviceArrayPointer_Raja(v);
  realtype *Jvdata      = N_VGetDeviceArrayPointer_Raja(Jv);

  const sunindextype zero = 0;

  N_VConst(ZERO, Jv);

  RAJA::forall<RAJA::cuda_exec<256> >(zero, NEQ, [=] __device__(sunindextype index) {
    sunindextype i = index/MY;
    sunindextype j = index%MY;

    Jvdata[index] = -TWO*(verdc+hordc) * vdata[index];
    if (i !=    0) Jvdata[index] += (hordc - horac) * vdata[index-MY];
    if (i != MX-1) Jvdata[index] += (hordc + horac) * vdata[index+MY];
    if (j !=    0) Jvdata[index] += verdc * vdata[index-1];
    if (j != MY-1) Jvdata[index] += verdc * vdata[index+1];
  });
  
  return(0);
}

/*
 *-------------------------------
 * Private helper functions
 *-------------------------------
 */

/* Print first lines of output (problem description) */

static void PrintHeader(realtype reltol, realtype abstol, realtype umax, UserData data)
{
  printf("\n2-D Advection-Diffusion Equation\n");
  printf("Mesh dimensions = %d X %d\n", data->MX, data->MY);
  printf("Total system size = %d\n", data->NEQ);
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("Tolerance parameters: reltol = %Lg   abstol = %Lg\n\n",
         reltol, abstol);
  printf("At t = %Lg      max.norm(u) =%14.6Le \n", T0, umax);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("Tolerance parameters: reltol = %g   abstol = %g\n\n",
         reltol, abstol);
  printf("At t = %g      max.norm(u) =%14.6e \n", T0, umax);
#else
  printf("Tolerance parameters: reltol = %g   abstol = %g\n\n", reltol, abstol);
  printf("At t = %g      max.norm(u) =%14.6e \n", T0, umax);
#endif

  return;
}

/* Print current value */

static void PrintOutput(realtype t, realtype umax, long int nst)
{
#if defined(SUNDIALS_EXTENDED_PRECISION)
  printf("At t = %4.2Lf   max.norm(u) =%14.6Le   nst = %4ld\n", t, umax, nst);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  printf("At t = %4.2f   max.norm(u) =%14.6e   nst = %4ld\n", t, umax, nst);
#else
  printf("At t = %4.2f   max.norm(u) =%14.6e   nst = %4ld\n", t, umax, nst);
#endif

  return;
}

/* Get and print some final statistics */

static void PrintFinalStats(void *cvode_mem)
{
  long lenrw, leniw ;
  long lenrwLS, leniwLS;
  long int nst, nfe, nsetups, nni, ncfn, netf;
  long int nli, npe, nps, ncfl, nfeLS;
  int flag;

  flag = CVodeGetWorkSpace(cvode_mem, &lenrw, &leniw);
  check_flag(&flag, "CVodeGetWorkSpace", 1);
  flag = CVodeGetNumSteps(cvode_mem, &nst);
  check_flag(&flag, "CVodeGetNumSteps", 1);
  flag = CVodeGetNumRhsEvals(cvode_mem, &nfe);
  check_flag(&flag, "CVodeGetNumRhsEvals", 1);
  flag = CVodeGetNumLinSolvSetups(cvode_mem, &nsetups);
  check_flag(&flag, "CVodeGetNumLinSolvSetups", 1);
  flag = CVodeGetNumErrTestFails(cvode_mem, &netf);
  check_flag(&flag, "CVodeGetNumErrTestFails", 1);
  flag = CVodeGetNumNonlinSolvIters(cvode_mem, &nni);
  check_flag(&flag, "CVodeGetNumNonlinSolvIters", 1);
  flag = CVodeGetNumNonlinSolvConvFails(cvode_mem, &ncfn);
  check_flag(&flag, "CVodeGetNumNonlinSolvConvFails", 1);

  flag = CVSpilsGetWorkSpace(cvode_mem, &lenrwLS, &leniwLS);
  check_flag(&flag, "CVSpilsGetWorkSpace", 1);
  flag = CVSpilsGetNumLinIters(cvode_mem, &nli);
  check_flag(&flag, "CVSpilsGetNumLinIters", 1);
  flag = CVSpilsGetNumPrecEvals(cvode_mem, &npe);
  check_flag(&flag, "CVSpilsGetNumPrecEvals", 1);
  flag = CVSpilsGetNumPrecSolves(cvode_mem, &nps);
  check_flag(&flag, "CVSpilsGetNumPrecSolves", 1);
  flag = CVSpilsGetNumConvFails(cvode_mem, &ncfl);
  check_flag(&flag, "CVSpilsGetNumConvFails", 1);
  flag = CVSpilsGetNumRhsEvals(cvode_mem, &nfeLS);
  check_flag(&flag, "CVSpilsGetNumRhsEvals", 1);

  printf("\nFinal Statistics.. \n\n");
  printf("lenrw   = %5ld     leniw   = %5ld\n", lenrw, leniw);
  printf("lenrwLS = %5ld     leniwLS = %5ld\n", lenrwLS, leniwLS);
  printf("nst     = %5ld\n"                  , nst);
  printf("nfe     = %5ld     nfeLS   = %5ld\n"  , nfe, nfeLS);
  printf("nni     = %5ld     nli     = %5ld\n"  , nni, nli);
  printf("nsetups = %5ld     netf    = %5ld\n"  , nsetups, netf);
  printf("npe     = %5ld     nps     = %5ld\n"  , npe, nps);
  printf("ncfn    = %5ld     ncfl    = %5ld\n\n", ncfn, ncfl);

  return;
}

/* Check function return value...
     opt == 0 means SUNDIALS function allocates memory so check if
              returned NULL pointer
     opt == 1 means SUNDIALS function returns a flag so check if
              flag >= 0
     opt == 2 means function allocates memory so check if returned
              NULL pointer */

static int check_flag(void *flagvalue, const char *funcname, int opt)
{
  int *errflag;

  /* Check if SUNDIALS function returned NULL pointer - no memory allocated */

  if (opt == 0 && flagvalue == NULL) {
    fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  /* Check if flag < 0 */

  else if (opt == 1) {
    errflag = (int *) flagvalue;
    if (*errflag < 0) {
      fprintf(stderr, "\nSUNDIALS_ERROR: %s() failed with flag = %d\n\n",
              funcname, *errflag);
      return(1); }}

  /* Check if function returned NULL pointer - no memory allocated */

  else if (opt == 2 && flagvalue == NULL) {
    fprintf(stderr, "\nMEMORY_ERROR: %s() failed - returned NULL pointer\n\n",
            funcname);
    return(1); }

  return(0);
}
